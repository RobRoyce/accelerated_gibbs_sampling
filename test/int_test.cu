#include "hip/hip_runtime.h"
#include <random>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include "../src/gmm.h"

static uint64_t usec;
static __inline__ uint64_t gettime(void) {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (((uint64_t) tv.tv_sec) * 1000000 + ((uint64_t) tv.tv_usec));
}
__attribute__ ((noinline))  void begin_roi() { usec = gettime(); }
__attribute__ ((noinline))  void end_roi() {
    usec = (gettime() - usec);
    printf("elapsed (sec): %f\n", usec / 1000000.0);
}

void printParams(struct GMMParams *params, DTYPE *data, size_t n, size_t k);

void randomInit(DTYPE *data, unsigned *zs, const int n, const int k);

void verify(struct GMMParams *params, unsigned *zs, size_t n);

#ifndef NSAMPLES
    #define NSAMPLES (16384)
#endif
#ifndef KCLASSES
    #define KCLASSES (16)
#endif
#ifndef MSAMPLERS
    #define MSAMPLERS (16)
#endif

int DEBUG = 1;
const int N = NSAMPLES;
const int K = KCLASSES;
const int M = MSAMPLERS; // Total independent Gibbs samplers
const int ITERS = 500;

const struct GMMPrior PRIOR = {
        .dirichletPrior=5.0,
        .meansMeanPrior=0.0,
        .meansVarPrior=100.0,
        .varsShapePrior=2.0,
        .varsScalePrior=10.0
};

int main(int argc, char **argv) {
    DEBUG = (argc > 1) && (strcmp(argv[1], "--debug") == 0) ? 1 : 0;
    srand(42);

    unsigned *h_zs = new unsigned[N];
    DTYPE *dataManaged = nullptr;
    struct GmmGibbsState *gibbsState = nullptr;

    const unsigned DATA_MEM_SIZE = N * sizeof(DTYPE);
    gpuErrchk(hipMallocManaged(&dataManaged, DATA_MEM_SIZE));

    // Synthesize data
    randomInit(dataManaged, h_zs, N, K);

    // Partition dataManaged into M subsets and allocate each subset to a unique GmmGibbsState struct.
    allocGmmGibbsState(&gibbsState, N, K, M, dataManaged, PRIOR);

    begin_roi();
    gibbs(gibbsState, M, ITERS);
    end_roi();

    for(int i = 0; i < M; i++)
    {
        printParams(gibbsState[i].params, gibbsState[i].data, gibbsState[i].n, gibbsState[i].k);
        verify(gibbsState[i].params, h_zs, N);
    }

    freeGmmGibbsState(gibbsState, M);
    gpuErrchk(hipFree(dataManaged));
    delete[] h_zs;
    return 0;
}

void printParams(struct GMMParams *params, DTYPE *data, size_t n, size_t k) {
    printf("%lu\n", k);
    for (int i = 0; i < k; i++)
        printf("%f %f %f\n", params->weights[i], params->means[i],
               params->vars[i]);
    for (int i = 0; i < n; i++)
        printf("%.2f ", data[i]);
    putchar('\n');
    for (int i = 0; i < n; i++)
        printf("%u ", params->zs[i]);
    putchar('\n');
}

void randomInit(DTYPE *data, unsigned *zs, const int n, const int k) {
    unsigned cat = 0;
    int min = 0, mod = 0;
    for (int i = 0; i < n; i++) {
        if (i < n / k) {
            min = 50;
            mod = 10;
            cat = 0;
        } else if (i < 2*n / k) {
            min = 12;
            mod = 4;
            cat = 1;
        } else if (i < 3*n / k) {
            min = -20;
            mod = 3;
            cat = 2;
        } else {
            min = -90;
            mod = 3;
            cat = 3;
        }
        data[i] = min + (rand() % mod);
        zs[i] = cat;
    }
}

void verify(struct GMMParams *params, unsigned *zs, size_t n) {
    int err = 0;

    for (int i = 0; i < N; i++) {
        if (params->zs[i] != zs[i]) {
            err = 1;
        }
    }

    if (err != 0)
        printf("int_test ---------------------------------------- FAILED! \n");
    else
        printf("int_test ---------------------------------------- SUCCESS! \n");
}