#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "gmm.h"
#include "utils.h"
#include "distrs.h"
#include "hip/hip_runtime.h"

__device__ struct gmm_gibbs_state *d_state;

struct gmm_sufficient_statistic {
    // `ns[i] = m` means m data points are assigned to the i-th component
    unsigned int *ns; 

    // sum of data points in each component
    double *comp_sums;

    // sum of the squares of data points in each component
    double *comp_sqsums;
};

struct gmm_gibbs_state {
    // number of data points
    size_t n;

    // number of mixture components
    size_t k; 

    double *data;

    struct gmm_prior prior;
    struct gmm_params *params;
    struct gmm_sufficient_statistic *ss;
};

struct gmm_gibbs_state *
alloc_gmm_gibbs_state(size_t n, size_t k, double *data, struct gmm_prior prior,
                      struct gmm_params *params)
{
    struct gmm_gibbs_state *s = (struct gmm_gibbs_state *)abort_malloc(sizeof(struct gmm_gibbs_state));
    s->n = n;
    s->k = k;
    s->data = data;
    s->prior = prior;
    s->params = params;
    s->ss = (struct gmm_sufficient_statistic*)
        abort_malloc(sizeof(struct gmm_sufficient_statistic));
    s->ss->ns = (unsigned *) abort_calloc(k, sizeof(unsigned int));
    s->ss->comp_sums = (double *) abort_calloc(k, sizeof(double));
    s->ss->comp_sqsums = (double *) abort_calloc(k, sizeof(double));
    return s;
}

void free_gmm_gibbs_state(struct gmm_gibbs_state *state)
{
    free(state->ss->ns);
    free(state->ss->comp_sums);
    free(state->ss->comp_sqsums);
    free(state->ss);
    free(state);
}

void clear_sufficient_statistic(struct gmm_gibbs_state *state)
{
    hipMemset(state->ss->ns, 0, state->k * sizeof(unsigned int));
    hipMemset(state->ss->comp_sums, 0, state->k * sizeof(double));
    hipMemset(state->ss->comp_sqsums, 0, state->k * sizeof(double));
}

__global__ void update_sufficient_statistic_cuda(struct gmm_gibbs_state *state)
{
    printf("Ok...\n");
    int i = threadIdx.x;
    double x = state->data[i];
    unsigned int z = state->params->zs[i];
    state->ss->ns[z]++;
    state->ss->comp_sums[z] += x;
    state->ss->comp_sqsums[z] += x*x;
}

void update_sufficient_statistic(struct gmm_gibbs_state *state)
{
    // XXX XXX this is the function that needs to be accelerated.
    clear_sufficient_statistic(state);
    for(size_t i=0; i < state->n; i++) {
        double x = state->data[i];
        unsigned int z = state->params->zs[i];
        state->ss->ns[z]++;
        state->ss->comp_sums[z] += x;
        state->ss->comp_sqsums[z] += x*x;
    }
}

void update_ws(struct gmm_gibbs_state *state)
{
    double dirichlet_param[state->k];
    vec_add_ud(dirichlet_param, state->ss->ns, state->params->weights, state->k);
    dirichlet(state->params->weights, dirichlet_param, state->k);
}

void update_means(struct gmm_gibbs_state *state)
{
    double k = 1/state->prior.means_var_prior,
           zeta = state->prior.means_mean_prior, mean, var;
    for(int j=0; j < state->k; j++) {
        double sum_xs = state->ss->comp_sums[j], ns = state->ss->ns[j],
               sigma2 = state->params->vars[j];
               mean = (k * zeta + sum_xs / sigma2) / (ns / sigma2 + k);
               var = 1/(ns / sigma2 + k);
        state->params->means[j] = gaussian(mean, var);
    }
}

void update_vars(struct gmm_gibbs_state *state)
{
    double alpha = state->prior.vars_shape_prior,
           beta = state->prior.vars_scale_prior, shape, scale;
    for(int j=0; j < state->k; j++) {
        double sum_xs = state->ss->comp_sums[j],
               sqsum_xs = state->ss->comp_sqsums[j],
               mu = state->params->means[j], ns = state->ss->ns[j];
               shape = alpha + ns/2;
               scale = beta + sqsum_xs/2 - mu*sum_xs + ns * mu*mu/2; 
        state->params->vars[j] = inverse_gamma(shape, scale);
    }
}

void update_zs(struct gmm_gibbs_state *state)
{
    double weights[state->k], mu, sigma2;
    for(int i=0; i < state->n; i++) {
        double x = state->data[i];
        for(int j=0; j < state->k; j++) {
            mu = state->params->means[j];
            sigma2 = state->params->vars[j];
            weights[j] = gaussian_pdf(x, mu, sigma2);
        }
        normalize(weights, state->k);
        state->params->zs[i] = categorical(weights, state->k);
    }
}

void gibbs(struct gmm_gibbs_state *state, size_t iters)
{
    int numBlocks = 1;
    dim3 threadsPerBlock(3);

    while(iters--) {
        printf("Clearing sufficient statistic...\n");
        clear_sufficient_statistic(state);

        struct gmm_gibbs_state *gibbs_state_d;
        hipMalloc((void**)&gibbs_state_d, sizeof(struct gmm_gibbs_state*));
        hipMemcpy(state, gibbs_state_d, sizeof(struct gmm_gibbs_state *), hipMemcpyHostToDevice);

        printf("Updating sufficient statistic...\n");
        update_sufficient_statistic_cuda<<<numBlocks, threadsPerBlock>>>(gibbs_state_d);

        hipMemcpy(gibbs_state_d, state, sizeof(struct gmm_gibbs_state *), hipMemcpyDeviceToHost);

        update_ws(state);
        update_means(state);
        update_vars(state);
        update_zs(state);
    }
}
