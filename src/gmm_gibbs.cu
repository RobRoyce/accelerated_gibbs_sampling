#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "gmm.h"
#include "utils.h"
#include "distrs.h"
#include "hip/hip_runtime.h"

__device__ struct gmm_gibbs_state *d_state;

struct gmm_sufficient_statistic {
    // `ns[i] = m` means m data points are assigned to the i-th component
    unsigned int *ns; 

    // sum of data points in each component
    double *comp_sums;

    // sum of the squares of data points in each component
    double *comp_sqsums;
};

struct gmm_gibbs_state {
    // number of data points
    size_t n;

    // number of mixture components
    size_t k; 

    double *data;

    struct gmm_prior prior;
    struct gmm_params *params;
    struct gmm_sufficient_statistic *ss;
};

struct gmm_gibbs_state *
alloc_gmm_gibbs_state(size_t n, size_t k, double *data, struct gmm_prior prior,
                      struct gmm_params *params)
{
    struct gmm_gibbs_state *s;
    hipMallocManaged(&s, sizeof(struct gmm_gibbs_state));
    
    s->n = n;
    s->k = k;
    s->data = data;
    s->prior = prior;
    s->params = params;

    hipMallocManaged(&(s->ss), sizeof(struct gmm_sufficient_statistic));
    hipMallocManaged(&(s->ss->ns), k*sizeof(unsigned int));
    hipMallocManaged(&(s->ss->comp_sums), k*sizeof(double));
    hipMallocManaged(&(s->ss->comp_sqsums), k*sizeof(double));

    // hipMemset(s->ss, 0, sizeof(struct gmm_sufficient_statistic));
    hipMemset(s->ss->ns, 0, k*sizeof(unsigned int));
    hipMemset(s->ss->comp_sums, 0, k*sizeof(double));
    hipMemset(s->ss->comp_sqsums, 0, k*sizeof(double));

    return s;
}

void free_gmm_gibbs_state(struct gmm_gibbs_state *state)
{
    hipFree(state->ss->ns);
    hipFree(state->ss->comp_sums);
    hipFree(state->ss->comp_sqsums);
    hipFree(state->ss);
    hipFree(state);
}

void clear_sufficient_statistic(struct gmm_gibbs_state *state)
{
    hipMemset(state->ss->ns, 0, state->k * sizeof(unsigned int));
    hipMemset(state->ss->comp_sums, 0, state->k * sizeof(double));
    hipMemset(state->ss->comp_sqsums, 0, state->k * sizeof(double));
}

__global__ void update_sufficient_statistic_cuda(struct gmm_gibbs_state *state)
{
    // printf("Ok...\n");
    int i = threadIdx.x;
    double x = state->data[i];
    unsigned int z = state->params->zs[i];

    // state->ss->ns[z]++;
    // state->ss->comp_sums[z] += x;
    // state->ss->comp_sqsums[z] += x*x;
    atomicAdd(&(state->ss->ns[z]), 1);
    atomicAdd(&(state->ss->comp_sums[z]), x);
    atomicAdd(&(state->ss->comp_sqsums[z]), x*x);
}

void update_sufficient_statistic(struct gmm_gibbs_state *state)
{
    // XXX XXX this is the function that needs to be accelerated.
    clear_sufficient_statistic(state);
    for(size_t i=0; i < state->n; i++) {
        double x = state->data[i];
        unsigned int z = state->params->zs[i];
        state->ss->ns[z]++;
        state->ss->comp_sums[z] += x;
        state->ss->comp_sqsums[z] += x*x;
    }
}

void update_ws(struct gmm_gibbs_state *state)
{
    // double dirichlet_param[state->k];
    double *dirichlet_param;
    hipMallocManaged(&dirichlet_param, state->k * sizeof(double));

    vec_add_ud(dirichlet_param, state->ss->ns, state->params->weights, state->k);
    dirichlet(state->params->weights, dirichlet_param, state->k);

    hipFree(dirichlet_param);
}

void update_means(struct gmm_gibbs_state *state)
{
    double k = 1/state->prior.means_var_prior,
           zeta = state->prior.means_mean_prior, mean, var;
    for(int j=0; j < state->k; j++) {
        double sum_xs = state->ss->comp_sums[j], ns = state->ss->ns[j],
               sigma2 = state->params->vars[j];
               mean = (k * zeta + sum_xs / sigma2) / (ns / sigma2 + k);
               var = 1/(ns / sigma2 + k);
        state->params->means[j] = gaussian(mean, var);
    }
}

void update_vars(struct gmm_gibbs_state *state)
{
    double alpha = state->prior.vars_shape_prior,
           beta = state->prior.vars_scale_prior, shape, scale;
    for(int j=0; j < state->k; j++) {
        double sum_xs = state->ss->comp_sums[j],
               sqsum_xs = state->ss->comp_sqsums[j],
               mu = state->params->means[j], ns = state->ss->ns[j];
               shape = alpha + ns/2;
               scale = beta + sqsum_xs/2 - mu*sum_xs + ns * mu*mu/2; 
        state->params->vars[j] = inverse_gamma(shape, scale);
    }
}

void update_zs(struct gmm_gibbs_state *state)
{
    double weights[state->k], mu, sigma2;
    for(int i=0; i < state->n; i++) {
        double x = state->data[i];
        for(int j=0; j < state->k; j++) {
            mu = state->params->means[j];
            sigma2 = state->params->vars[j];
            weights[j] = gaussian_pdf(x, mu, sigma2);
        }
        normalize(weights, state->k);
        state->params->zs[i] = categorical(weights, state->k);
    }
}

void gibbs(struct gmm_gibbs_state *state, size_t iters)
{
    int numBlocks = 1;
    dim3 threadsPerBlock(state->n);

    while(iters--) {

        clear_sufficient_statistic(state);
        update_sufficient_statistic_cuda<<<numBlocks, threadsPerBlock>>>(state);
        hipDeviceSynchronize();

        update_ws(state);
        update_means(state);
        update_vars(state);
        update_zs(state);
    }
}
