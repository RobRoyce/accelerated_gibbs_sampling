#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "gmm.h"
#include "distrs.h"

#ifndef NSAMPLES
    #define NSAMPLES (1024)
#endif
#ifndef KCLASSES
    #define KCLASSES (16)
#endif
#ifndef MSAMPLERS
    #define MSAMPLERS (4)
#endif

// cuRAND state array for uniform distributions
__device__ hiprandState curandStates[MSAMPLERS];

__global__ void setup_kernel() {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(1234, id, 0, &(curandStates[id]));
}

inline void swap(DTYPE *data, int i, int j)
{
    DTYPE tmp = data[i];
    data[i] = data[j];
    data[j] = tmp;
}

void shuffle(DTYPE *data, int n)
{
    for(int i = 0; i < n; i++)
    {
        unsigned int swapIdx = (unsigned int)rand() % n;
        swap(data, i, swapIdx);
    }
}

void allocGmmGibbsState(struct GmmGibbsState **s, size_t n, size_t k, size_t m, DTYPE *data, struct GMMPrior prior) {

    gpuErrchk(hipMallocManaged(s, m * sizeof(struct GmmGibbsState)));

    // Shuffle the dataset such that each independent sampler gets a decent representation of the problem
    shuffle(data, n);

    struct GmmGibbsState *state;
    for(int i = 0; i < m; i++)
    {    
        const unsigned CLASS_MEM_SIZE = k * sizeof(DTYPE),
                PARAM_MEM_SIZE = sizeof(struct GMMParams),
                ZS_MEM_SIZE = n * sizeof(unsigned);
        struct GMMParams *params = nullptr;

        gpuErrchk(hipMallocManaged(&params, PARAM_MEM_SIZE));
        gpuErrchk(hipMallocManaged(&(params->weights), CLASS_MEM_SIZE));
        gpuErrchk(hipMallocManaged(&(params->means), CLASS_MEM_SIZE));
        gpuErrchk(hipMallocManaged(&(params->vars), CLASS_MEM_SIZE));
        gpuErrchk(hipMallocManaged(&(params->zs), ZS_MEM_SIZE));

        state = &(*s)[i];
        state->n = n/m;
        state->k = k;

        state->data = &data[state->n * i];
        state->prior = prior;

        randInitGmmParams(params, n, k, prior);

        state->params = params;

        gpuErrchk(hipMallocManaged(&(state->ss), sizeof(struct GmmSufficientStatistic)));
        gpuErrchk(hipMallocManaged(&(state->ss->ns), k * sizeof(unsigned int)));
        gpuErrchk(hipMallocManaged(&(state->ss->compSums), k * sizeof(DTYPE)));
        gpuErrchk(hipMallocManaged(&(state->ss->compSquaredSums), k * sizeof(DTYPE)));

        // hipMemset(state->ss, 0, sizeof(struct GmmSufficientStatistic));
        gpuErrchk(hipMemset(state->ss->ns, 0, k * sizeof(unsigned int)));
        gpuErrchk(hipMemset(state->ss->compSums, 0, k * sizeof(DTYPE)));
        gpuErrchk(hipMemset(state->ss->compSquaredSums, 0, k * sizeof(DTYPE)));
    }
}

void freeGmmGibbsState(struct GmmGibbsState *state, size_t m) {
    for (int i=0; i < m; i++)
    {
        struct GmmGibbsState *s = &state[i];
        gpuErrchk(hipFree(s->ss->ns));
        gpuErrchk(hipFree(s->ss->compSums));
        gpuErrchk(hipFree(s->ss->compSquaredSums));
        gpuErrchk(hipFree(s->ss));
        gpuErrchk(hipFree(s));
    }
}

__device__ void clearSufficientStatistic(struct GmmGibbsState *state) {
    for(int i = 0; i < state->k; i++)
    {
        state->ss->ns[i] = 0;
        state->ss->compSums[i] = 0;
        state->ss->compSquaredSums[i] = 0;
    }
}

__device__ void updateSufficientStatistic(struct GmmGibbsState *state) {
    for(size_t i=0; i < state->n; i++) {
        DTYPE x = state->data[i];
        unsigned int z = state->params->zs[i];
        state->ss->ns[z]++;
        state->ss->compSums[z] += x;
        state->ss->compSquaredSums[z] += x*x;
    }
}

__device__ void updateWeights(struct GmmGibbsState *state) {

    DTYPE dirichlet_param[KCLASSES];
    vecAddUd(dirichlet_param, state->ss->ns, state->params->weights, state->k);
    dirichlet(state->params->weights, dirichlet_param, state->k);

}

__device__ void updateMeans(struct GmmGibbsState *state) {
    DTYPE k = 1/state->prior.meansVarPrior,
           zeta = state->prior.meansMeanPrior, mean, var;
    for(int j=0; j < state->k; j++) {
        DTYPE sum_xs = state->ss->compSums[j], ns = state->ss->ns[j],
               sigma2 = state->params->vars[j];
               mean = (k * zeta + sum_xs / sigma2) / (ns / sigma2 + k);
               var = 1/(ns / sigma2 + k);
        state->params->means[j] = gaussian(mean, var);
    }
}

__device__ void updateVars(struct GmmGibbsState *state) {
    DTYPE alpha = state->prior.varsShapePrior,
           beta = state->prior.varsScalePrior, shape, scale;
    for(int j=0; j < state->k; j++) {
        DTYPE sum_xs = state->ss->compSums[j],
               sqsum_xs = state->ss->compSquaredSums[j],
               mu = state->params->means[j], ns = state->ss->ns[j];
               shape = alpha + ns/2;
               scale = beta + sqsum_xs/2 - mu*sum_xs + ns * mu*mu/2; 
        state->params->vars[j] = inverse_gamma(shape, scale);
    }
}

__device__ void updateZs(struct GmmGibbsState *state) {
    DTYPE weights[KCLASSES], mu, sigma2;
    for(int i=0; i < state->n; i++) {
        DTYPE x = state->data[i];
        for(int j=0; j < state->k; j++) {
            mu = state->params->means[j];
            sigma2 = state->params->vars[j];
            weights[j] = gaussian_pdf(x, mu, sigma2);
        }
        normalize(weights, state->k);
        state->params->zs[i] = categorical(weights, state->k);
    }
}

__global__ void gibbsCuda(struct GmmGibbsState *gibbsStates, size_t iters) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    struct GmmGibbsState *state = &gibbsStates[i];

    while (iters--) {
        clearSufficientStatistic(state);
        updateSufficientStatistic(state);
        updateWeights(state);
        updateMeans(state);
        updateVars(state);
        updateZs(state);
    }

}

void gibbs(struct GmmGibbsState *gibbsStates, int num_states, size_t iters) {
    
    if(num_states < 32)
    {
        // Initialize CUDA random states
        setup_kernel<<<num_states, 1>>>();

        // Run independent Gibbs samplers
        gibbsCuda<<<num_states, 1>>>(gibbsStates, iters);
    }
    else
    {
        // Initialize CUDA random states
        setup_kernel<<<32, num_states/32>>>();

        // Run independent Gibbs samplers
        gibbsCuda<<<32, num_states/32>>>(gibbsStates, iters);
    }

    gpuErrchk(hipDeviceSynchronize());
}
