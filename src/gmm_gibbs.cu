#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "gmm.h"
#include "distrs.h"

#ifndef NSAMPLES
    #define NSAMPLES (1024)
#endif
#ifndef KCLASSES
    #define KCLASSES (16)
#endif

// cuRAND state array for uniform distributions
__device__ hiprandState curandStates[NSAMPLES];

__global__ void setup_kernel() {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(1234, id, 0, &(curandStates[id]));
}

void allocGmmGibbsState(struct GmmGibbsState **s, size_t n, size_t k, DTYPE *data,
                        struct GMMPrior prior, struct GMMParams *params) {
    gpuErrchk(hipMallocManaged(s, sizeof(struct GmmGibbsState)));

    struct GmmGibbsState *state = *s;
    state->n = n;
    state->k = k;

    state->data = data;
    state->prior = prior;
    state->params = params;

    gpuErrchk(hipMallocManaged(&(state->ss), sizeof(struct GmmSufficientStatistic)));
    gpuErrchk(hipMallocManaged(&(state->ss->ns), k * sizeof(unsigned int)));
    gpuErrchk(hipMallocManaged(&(state->ss->compSums), k * sizeof(DTYPE)));
    gpuErrchk(hipMallocManaged(&(state->ss->compSquaredSums), k * sizeof(DTYPE)));

    // hipMemset(state->ss, 0, sizeof(struct GmmSufficientStatistic));
    gpuErrchk(hipMemset(state->ss->ns, 0, k * sizeof(unsigned int)));
    gpuErrchk(hipMemset(state->ss->compSums, 0, k * sizeof(DTYPE)));
    gpuErrchk(hipMemset(state->ss->compSquaredSums, 0, k * sizeof(DTYPE)));
}

void freeGmmGibbsState(struct GmmGibbsState *state) {
    gpuErrchk(hipFree(state->ss->ns));
    gpuErrchk(hipFree(state->ss->compSums));
    gpuErrchk(hipFree(state->ss->compSquaredSums));
    gpuErrchk(hipFree(state->ss));
    gpuErrchk(hipFree(state));
}

__device__ void clearSufficientStatistic(struct GmmGibbsState *state) {
    for(int i = 0; i < state->k; i++)
    {
        state->ss->ns[i] = 0;
        state->ss->compSums[i] = 0;
        state->ss->compSquaredSums[i] = 0;
    }
}

__device__ void updateSufficientStatistic(struct GmmGibbsState *state) {
    for(size_t i=0; i < state->n; i++) {
        DTYPE x = state->data[i];
        unsigned int z = state->params->zs[i];
        state->ss->ns[z]++;
        state->ss->compSums[z] += x;
        state->ss->compSquaredSums[z] += x*x;
    }
}

__device__ void updateWeights(struct GmmGibbsState *state) {

    DTYPE dirichlet_param[KCLASSES];
    vecAddUd(dirichlet_param, state->ss->ns, state->params->weights, state->k);
    dirichlet(state->params->weights, dirichlet_param, state->k);

}

__device__ void updateMeans(struct GmmGibbsState *state) {
    DTYPE k = 1/state->prior.meansVarPrior,
           zeta = state->prior.meansMeanPrior, mean, var;
    for(int j=0; j < state->k; j++) {
        DTYPE sum_xs = state->ss->compSums[j], ns = state->ss->ns[j],
               sigma2 = state->params->vars[j];
               mean = (k * zeta + sum_xs / sigma2) / (ns / sigma2 + k);
               var = 1/(ns / sigma2 + k);
        state->params->means[j] = gaussian(mean, var);
    }
}

__device__ void updateVars(struct GmmGibbsState *state) {
    DTYPE alpha = state->prior.varsShapePrior,
           beta = state->prior.varsScalePrior, shape, scale;
    for(int j=0; j < state->k; j++) {
        DTYPE sum_xs = state->ss->compSums[j],
               sqsum_xs = state->ss->compSquaredSums[j],
               mu = state->params->means[j], ns = state->ss->ns[j];
               shape = alpha + ns/2;
               scale = beta + sqsum_xs/2 - mu*sum_xs + ns * mu*mu/2; 
        state->params->vars[j] = inverse_gamma(shape, scale);
    }
}

__device__ void updateZs(struct GmmGibbsState *state) {
    DTYPE weights[KCLASSES], mu, sigma2;
    for(int i=0; i < state->n; i++) {
        DTYPE x = state->data[i];
        for(int j=0; j < state->k; j++) {
            mu = state->params->means[j];
            sigma2 = state->params->vars[j];
            weights[j] = gaussian_pdf(x, mu, sigma2);
        }
        normalize(weights, state->k);
        state->params->zs[i] = categorical(weights, state->k);
    }
}

__global__ void gibbsCuda(struct GmmGibbsState *gibbsStates, size_t iters) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    while (iters--) {
        clearSufficientStatistic(&gibbsStates[i]);
        updateSufficientStatistic(&gibbsStates[i]);
        updateWeights(&gibbsStates[i]);
        updateMeans(&gibbsStates[i]);
        updateVars(&gibbsStates[i]);
        updateZs(&gibbsStates[i]);
    }


}

void gibbs(struct GmmGibbsState *gibbsStates, int num_states, size_t iters) {
    dim3 nThreads(1024, 1, 1);
    dim3 nBlocks(gibbsStates->n / nThreads.x, 1, 1);

    // Initialize CUDA random states
    setup_kernel<<<1024, (gibbsStates->n * num_states) / 1024>>>();

    // Run independent Gibbs samplers
    // gibbsCuda<<<32, num_states/32>>>(gibbsStates, iters);
    gibbsCuda<<<1, 1>>>(gibbsStates, iters);
    gpuErrchk(hipDeviceSynchronize());
}
