
#include <hip/hip_runtime.h>
#include <float.h>
#include <math.h>
#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>


void vec_add_dd(double *dst, double *u, double *v, size_t k)
{
    for(int i=0; i < k; i++)
        dst[i] = u[i] + v[i];
}

void vec_add_ud(double *dst, unsigned int *u, double *v, size_t k)
{
    for(int i=0; i < k; i++)
        dst[i] = u[i] + v[i];
}

void normalize(double *v, size_t n)
{
    double sum = 0;
    for(int i=0; i < n; i++)
        sum += v[i];
    for(int i=0; i < n; i++)
        v[i] /= sum;
}

double square(double x) { return x*x; }

double ligamma(double s, double x)
{
    double sum=0, term=1/s;
    for(int k=1; term >= DBL_EPSILON; sum += term, term *= x/(s + k), k++);
    return pow(x, s) * exp(-x) * sum; 
}

double uigamma(double s, double x) { return tgamma(s) - ligamma(s, x); }

double beta(double *x, size_t n)
{
    double gamma_prod=1, sum=0;
    for(int i=0; i < n; gamma_prod *= tgamma(x[i]), sum += x[i], i++);
    return gamma_prod / tgamma(sum);
}

void *abort_calloc(size_t nmemb, size_t size)
{
    void *mem = calloc(nmemb, size);
    if(mem == NULL)
        fputs("insufficient memory", stderr), abort();
    return mem;
}

void *abort_malloc(size_t size) { return abort_calloc(1, size); }
